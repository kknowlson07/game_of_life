#include "hip/hip_runtime.h"
#include "mpi_life.hpp"

__global__ void updateKernel(unsigned long dim, char* A, char* B)
{
    int row = blockIdx.x*blockDim.x + threadIdx.x +1;
    int col = blockIdx.y*blockDim.y + threadIdx.y +1;


    //consider making a 1-dim block and let thread iterate across a single row of matrix?

    if (row < dim +1  && col < dim +1 && row > 0 && col > 0){
        unsigned long n;
        unsigned long padDim = dim + 2;
        int liveNeighbors;


        n = (row * (dim+2)) + col;

        liveNeighbors = A[n - 1] +
            A[n + 1] +
            A[n - (padDim)] +
            A[n + (padDim)] +
            A[n - (padDim) - 1]+
            A[n + (padDim) - 1]+
            A[n - (padDim) + 1]+
            A[n + (padDim) + 1];

        // alive
        if (A[n]) {
            if (liveNeighbors < 2 || liveNeighbors > 3) {
                B[n] = 0; // dead due to underpopulation or overpopulation
            } else {
                B[n] = 1; // survive to the next generation
            }
        } else { // dead
            if (liveNeighbors == 3) {
                B[n] = 1; // becomes alive due to reproduction
            } else {
                B[n] = 0;   // stay dead
            }
        }

    }
}

